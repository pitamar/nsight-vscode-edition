/*

The MIT License (MIT)

Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a copy of
this software and associated documentation files (the "Software"), to deal in
the Software without restriction, including without limitation the rights to
use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
the Software, and to permit persons to whom the Software is furnished to do so,
subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

*/

#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>

#include <hip/hip_runtime.h>

#if defined(assert)
#undef assert
#endif

#define assert(c) \
    do { \
        if(!(c)) { \
            fprintf(stderr, "Assertion \"%s\" failed. (%s:%d)\n", \
                #c, __FILE__, __LINE__); \
            exit(1); \
        } \
    } while(0)

#define assertSucceeded(c) \
    do { \
        unsigned __tmp = c; \
        if(__tmp != hipSuccess) { \
            fprintf(stderr, "Operation \"%s\" failed with error code %x. (%s:%d)\n", \
                #c, (__tmp), __FILE__, __LINE__); \
            exit(__tmp); \
        } \
    } while(0)

#define ARRAY_LENGTH(x) (sizeof(x) / sizeof(x[0]))

constexpr int dataLength = 1 << 5;
constexpr int threadsPerBlock = 1 << 5;

typedef unsigned char byte;

struct TestType
{
    union {
        struct
        {
            unsigned lowHalf;
            unsigned highHalf;
        } halfAndHalf;

        unsigned long long whole;
    } takeYourPick;

    int arr[5];

    struct {
        char a;
        char b;
    } structArr[5];

    float theFloats[2];
    double theDouble;
};

__device__ void cudaComputeHashInner(TestType* input, unsigned *results)
{
    int idx = blockIdx.x * threadsPerBlock + threadIdx.x;
    TestType* myInput = input + idx;

    unsigned myResult = 0;

    myResult += myInput->takeYourPick.halfAndHalf.lowHalf - idx;
    myResult += myInput->takeYourPick.halfAndHalf.highHalf - idx;

    for(size_t i = 0; i < ARRAY_LENGTH(myInput->arr); i++)
    {
        myResult += myInput->arr[i] - idx;
    }

    for(size_t i = 0; i < sizeof(myInput->structArr); i++)
    {
        myResult += reinterpret_cast<byte *>(myInput->structArr)[i] - '0';
    }

    __syncthreads();

    results[idx] = myResult;
}

__global__ void cudaComputeHash(TestType* input, unsigned *results)
{
    int idx = blockIdx.x * threadsPerBlock + threadIdx.x;
    cudaComputeHashInner(input, results);
    results[idx] += 1;
}

int main()
{
    int cudaDeviceCount;
    assertSucceeded(hipGetDeviceCount(&cudaDeviceCount));
    assert(cudaDeviceCount > 0);

    assertSucceeded(hipSetDevice(0));

    TestType* input;
    unsigned* results;

    assertSucceeded(hipMallocManaged(&input, sizeof(TestType) * dataLength));
    assert(!!input);

    for (size_t i = 0; i < dataLength; i++)
    {
        input[i].takeYourPick.halfAndHalf.lowHalf = i + 1;
        input[i].takeYourPick.halfAndHalf.highHalf = i + 3;

        for(size_t j = 0; j < ARRAY_LENGTH(input[i].arr); j++)
        {
            input[i].arr[j] = i + j + 2;
        }

        for(size_t j = 0; j < sizeof(input[i].structArr); j++)
        {
            reinterpret_cast<byte *>(input[i].structArr)[j] = '0' + static_cast<char>((i + j) % 10);
        }

        input[i].theFloats[0] = i + 1;
        input[i].theFloats[1] = input[i].theFloats[0] / 2;

        input[i].theDouble = input[i].theFloats[1] + 1;
    }

    assertSucceeded(hipMallocManaged(reinterpret_cast<void **>(&results), sizeof(unsigned) * dataLength));
    assert(!!results);

    constexpr int blocks = dataLength / threadsPerBlock;
    cudaComputeHash<<<blocks, threadsPerBlock>>>(input, results);

    assertSucceeded(hipDeviceSynchronize());

    const unsigned expectedResult =
        1 +
        3 +
        ARRAY_LENGTH(input[0].arr) * (ARRAY_LENGTH(input[0].arr) - 1) / 2 +
        ARRAY_LENGTH(input[0].arr) * 2 +
        sizeof(input[0].structArr) * (sizeof(input[0].structArr) - 1) / 2 +
        1; // Added by cudaComputeHash (rather than by cudaComputeHashInner)

    for (unsigned i = 0; i < dataLength; i++)
    {
        if (results[i] != expectedResult){
            fprintf(stderr, "results[%u] (%u) != %u\n", i, results[i], expectedResult);
            exit(1);
        }
    }

    assertSucceeded(hipFree(input));
    assertSucceeded(hipFree(results));

    fprintf(stderr, "Success\n");

    exit(0);
}
